#include "hip/hip_runtime.h"
#include "foo.h"

#include <cstdio>
#include <iostream>
#include <stdexcept>

__global__ void fooKernel()
{
    printf("Running fooKernel\n");
}

void foo()
{
    fooKernel<<<1,1>>>();
    hipError_t cuda_result_ = hipPeekAtLastError();
    if (cuda_result_ != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(cuda_result_));
    }
    hipDeviceSynchronize();
}